#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>

__global__ void mandelKernel(float* dlowerX, float* dlowerY, float* dstepX, float* dstepY, int *dimg) {
    // To avoid error caused by the floating number, use the following pseudo code
    //
    // float x = lowerX + thisX * stepX;
    // float y = lowerY + thisY * stepY;

    // add by myself
    float x = *dlowerX + (blockIdx.x * blockDim.x + threadIdx.x) * *dstepX;
    float y = *dlowerY + (blockIdx.y * blockDim.y + threadIdx.y) * *dstepY;
    //
}

// Host front-end function that allocates the memory and launches the GPU kernel
void hostFE (float upperX, float upperY, float lowerX, float lowerY, int* img, int resX, int resY, int maxIterations)
{
    float stepX = (upperX - lowerX) / resX;
    float stepY = (upperY - lowerY) / resY;

    // add by myself
    int *temp_himg = (int*) malloc(sizeof(int) * resX * resY), *dimg;
    float *dlowerX, *dlowerY, *dstepX, *dstepY;
    hipMalloc(&dimg, sizeof(int) * resX * resY);
    hipMalloc(&dlowerX, sizeof(float));
    hipMalloc(&dlowerY, sizeof(float));
    hipMalloc(&dstepX, sizeof(float));
    hipMalloc(&dstepY, sizeof(float));
    hipMemcpy(dlowerX, &lowerX, sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(dlowerY, &lowerY, sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(dstepX, &stepX, sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(dstepY, &stepY, sizeof(float), hipMemcpyHostToDevice);



    hipMemcpy(dimg, temp_himg, sizeof(int) * resX * resY, hipMemcpyDeviceToHost);
    hipFree(dimg); hipFree(dlowerX); hipFree(dlowerY); hipFree(dstepX); hipFree(dstepY);
    //
}
