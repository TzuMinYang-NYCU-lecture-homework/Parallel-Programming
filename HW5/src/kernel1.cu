#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>

#define BLOCK_WITDH 16

// add by myself
// copy from serial
__device__ int mandel(float x, float y, int maxIterations)
{
    float z_re = x, z_im = y;
    int k;

    for (k = 0; k < maxIterations; ++k)
    {
        if (z_re * z_re + z_im * z_im > 4.f) break;

        float new_re = z_re * z_re - z_im * z_im;
        float new_im = 2.f * z_re * z_im;
        z_re = x + new_re;
        z_im = y + new_im;
    }

    return k;
}
//

__global__ void mandelKernel(int* dimg, float lowerX, float lowerY, float stepX, float stepY, int resX, int maxIterations) {
    // To avoid error caused by the floating number, use the following pseudo code
    //
    // float x = lowerX + thisX * stepX;
    // float y = lowerY + thisY * stepY;

    // add by myself
    // indexing, x is horizontal, y is vertical
    int i = blockIdx.x * blockDim.x + threadIdx.x, j = blockIdx.y * blockDim.y + threadIdx.y;

    float x = lowerX + i * stepX;
    float y = lowerY + j * stepY;
    dimg[j * resX + i] = mandel(x, y, maxIterations);
    //
}

// Host front-end function that allocates the memory and launches the GPU kernel
void hostFE (float upperX, float upperY, float lowerX, float lowerY, int* img, int resX, int resY, int maxIterations)
{
    float stepX = (upperX - lowerX) / resX;
    float stepY = (upperY - lowerY) / resY;

    // add by myself
    int *temp_himg = (int*) malloc(sizeof(int) * resX * resY); // because of hw require

    // declare gpu var.
    int *dimg;

    // allocate gpu memory
    hipMalloc(&dimg, sizeof(int) * resX * resY);

    // call gpu kernel func.
    dim3 dimGrid(resX / BLOCK_WITDH, resY / BLOCK_WITDH), imBlock(BLOCK_WITDH, BLOCK_WITDH);
    mandelKernel<<<dimGrid, imBlock>>>(dimg, lowerX, lowerY, stepX, stepY, resX, maxIterations);

    // copy ans from gpu to host
    hipMemcpy(temp_himg, dimg, sizeof(int) * resX * resY, hipMemcpyDeviceToHost);

    // copy data to result (because of hw require)
    memcpy(img, temp_himg, sizeof(int) * resX * resY);

    // free gpu memory
    hipFree(dimg);
    //
}
