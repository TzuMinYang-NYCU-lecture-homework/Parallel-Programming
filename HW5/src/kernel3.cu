#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>

#define BLOCK_WITDH 8
#define TILE_WIDTH 2

// add by myself
// copy from serial
__device__ int mandel(float x, float y, int maxIterations)
{
    float z_re = x, z_im = y;
    int k;

    for (k = 0; k < maxIterations; ++k)
    {
        if (z_re * z_re + z_im * z_im > 4.f) break;

        float new_re = z_re * z_re - z_im * z_im;
        float new_im = 2.f * z_re * z_im;
        z_re = x + new_re;
        z_im = y + new_im;
    }

    return k;
}
//

__global__ void mandelKernel(size_t* dpitch, float* dlowerX, float* dlowerY, float* dstepX, float* dstepY, int* dresX, int* dimg, int* dmaxIterations) {
    // To avoid error caused by the floating number, use the following pseudo code
    //
    // float x = lowerX + thisX * stepX;
    // float y = lowerY + thisY * stepY;

    // add by myself
    // indexing, x is horizontal, y is vertical
    int i_start = (blockIdx.x * blockDim.x + threadIdx.x) * TILE_WIDTH, j_start = (blockIdx.y * blockDim.y + threadIdx.y) * TILE_WIDTH;

    for (int j = j_start; j < j_start + TILE_WIDTH; ++j)
    {
        for (int i = i_start; i < i_start + TILE_WIDTH; ++i)
        {
            float x = *dlowerX + i * *dstepX;
            float y = *dlowerY + j * *dstepY;
            // use this index because pitch memory
            // T* pElement = (T*)((char*)BaseAddress + Row * pitch) + Column;
            *((int*)((char*)dimg + j * *dpitch) + i) = mandel(x, y, *dmaxIterations);
        }
    }
    //
}

// Host front-end function that allocates the memory and launches the GPU kernel
void hostFE (float upperX, float upperY, float lowerX, float lowerY, int* img, int resX, int resY, int maxIterations)
{
    float stepX = (upperX - lowerX) / resX;
    float stepY = (upperY - lowerY) / resY;

    // add by myself
    int *temp_himg; // because of hw require

    // allocate pinned-page host memory (because of hw require)
    // __host__​cudaError_t cudaHostAlloc ( void** pHost, size_t size, unsigned int  flags )
    hipHostAlloc((void**) &temp_himg, sizeof(int) * resX * resY, hipHostMallocDefault);

    // declare gpu var.
    int *dimg, *dmaxIterations, *dresX;
    float *dlowerX, *dlowerY, *dstepX, *dstepY;

    // allocate pitch gpu memory (because of hw require), it will align 256 or 512, fast for hardware, useful for 2D or 3D picture
    size_t pitch, *dpitch;
    // __host__​cudaError_t cudaMallocPitch ( void** devPtr, size_t* pitch, size_t width, size_t height )
    hipMallocPitch((void**) &dimg, &pitch, sizeof(int) * resX, resY);

    // allocate gpu memory
    hipMalloc(&dpitch, sizeof(size_t));
    hipMalloc(&dlowerX, sizeof(float));
    hipMalloc(&dlowerY, sizeof(float));
    hipMalloc(&dstepX, sizeof(float));
    hipMalloc(&dstepY, sizeof(int));
    hipMalloc(&dstepY, sizeof(int));
    hipMalloc(&dresX, sizeof(int));
    hipMalloc(&dmaxIterations, sizeof(int));

    // copy data from host to gpu
    hipMemcpy(dpitch, &pitch, sizeof(size_t), hipMemcpyHostToDevice);
    hipMemcpy(dlowerX, &lowerX, sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(dlowerY, &lowerY, sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(dstepX, &stepX, sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(dstepY, &stepY, sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(dresX, &resX, sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(dmaxIterations, &maxIterations, sizeof(int), hipMemcpyHostToDevice);

    // call gpu kernel func.
    dim3 dimGrid(resX / BLOCK_WITDH / TILE_WIDTH, resY / BLOCK_WITDH / TILE_WIDTH), dimBlock(BLOCK_WITDH, BLOCK_WITDH); // block num(dimGrid) should reduce because of tile, one thread will deal with a tile of pixels
    mandelKernel<<<dimGrid, dimBlock>>>(dpitch, dlowerX, dlowerY, dstepX, dstepY, dresX, dimg, dmaxIterations);

    // copy ans from gpu to host, pitch memory can't use "cudaMemcpy"
    // __host__ cudaError_t cudaMemcpy2D(void *dst, size_t dpitch, const void *src, size_t spitch, size_t width, size_t height, enum cudaMemcpyKind kind);
    hipMemcpy2D(temp_himg, sizeof(int) * resX, dimg, pitch, sizeof(int) * resX, resY, hipMemcpyDeviceToHost);

    // copy data to result (because of hw require)
    memcpy(img, temp_himg, sizeof(int) * resX * resY);

    // free gpu memory
    hipHostFree(temp_himg);
    hipFree(dimg); hipFree(dlowerX); hipFree(dlowerY); 
    hipFree(dstepX); hipFree(dstepY); hipFree(dresX); hipFree(dmaxIterations);
    //
}
