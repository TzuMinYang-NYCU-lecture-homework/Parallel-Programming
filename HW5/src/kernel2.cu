#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>

#define BLOCK_WITDH 16

// add by myself
__device__ int mandel(float x, float y, int maxIterations)
{
    float z_re = x, z_im = y;
    int k;

    for (k = 0; k < maxIterations; ++k)
    {
        if (z_re * z_re + z_im * z_im > 4.f) break;

        float new_re = z_re * z_re - z_im * z_im;
        float new_im = 2.f * z_re * z_im;
        z_re = x + new_re;
        z_im = y + new_im;
    }

    return k;
}
//

__global__ void mandelKernel(size_t* dpitch, float* dlowerX, float* dlowerY, float* dstepX, float* dstepY, int* dresX, int* dimg, int* dmaxIterations) {
    // To avoid error caused by the floating number, use the following pseudo code
    //
    // float x = lowerX + thisX * stepX;
    // float y = lowerY + thisY * stepY;

    // add by myself
    int i = blockIdx.x * blockDim.x + threadIdx.x, j = blockIdx.y * blockDim.y + threadIdx.y;

    float x = *dlowerX + i * *dstepX;
    float y = *dlowerY + j * *dstepY;
    // T* pElement = (T*)((char*)BaseAddress + Row * pitch) + Column;
    *((int*)((char*)dimg + j * *dpitch) + i) = mandel(x, y, *dmaxIterations);
    //
}

// Host front-end function that allocates the memory and launches the GPU kernel
void hostFE (float upperX, float upperY, float lowerX, float lowerY, int* img, int resX, int resY, int maxIterations)
{
    float stepX = (upperX - lowerX) / resX;
    float stepY = (upperY - lowerY) / resY;

    // add by myself
    int *temp_himg;
    // __host__​cudaError_t cudaHostAlloc ( void** pHost, size_t size, unsigned int  flags )
    hipHostAlloc((void**) &temp_himg, sizeof(int) * resX * resY, hipHostMallocDefault);

    int *dimg, *dmaxIterations, *dresX;
    float *dlowerX, *dlowerY, *dstepX, *dstepY;

    size_t pitch, *dpitch;
    // __host__​cudaError_t cudaMallocPitch ( void** devPtr, size_t* pitch, size_t width, size_t height )
    hipMallocPitch((void**) &dimg, &pitch, sizeof(int) * resX, resY);
    hipMalloc(&dpitch, sizeof(size_t));
    hipMalloc(&dlowerX, sizeof(float));
    hipMalloc(&dlowerY, sizeof(float));
    hipMalloc(&dstepX, sizeof(float));
    hipMalloc(&dstepY, sizeof(int));
    hipMalloc(&dstepY, sizeof(int));
    hipMalloc(&dresX, sizeof(int));
    hipMalloc(&dmaxIterations, sizeof(int));

    hipMemcpy(dpitch, &pitch, sizeof(size_t), hipMemcpyHostToDevice);
    hipMemcpy(dlowerX, &lowerX, sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(dlowerY, &lowerY, sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(dstepX, &stepX, sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(dstepY, &stepY, sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(dresX, &resX, sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(dmaxIterations, &maxIterations, sizeof(int), hipMemcpyHostToDevice);

    dim3 dimGrid(resX / BLOCK_WITDH, resY / BLOCK_WITDH), dimBlock(BLOCK_WITDH, BLOCK_WITDH);
    mandelKernel<<<dimGrid, dimBlock>>>(dpitch, dlowerX, dlowerY, dstepX, dstepY, dresX, dimg, dmaxIterations);

    // __host__ cudaError_t cudaMemcpy2D(void *dst, size_t dpitch, const void *src, size_t spitch, size_t width, size_t height, enum cudaMemcpyKind kind);
    hipMemcpy2D(temp_himg, sizeof(int) * resX, dimg, pitch, sizeof(int) * resX, resY, hipMemcpyDeviceToHost);

    memcpy(img, temp_himg, sizeof(int) * resX * resY);

    hipHostFree(temp_himg);
    hipFree(dimg); hipFree(dlowerX); hipFree(dlowerY); 
    hipFree(dstepX); hipFree(dstepY); hipFree(dresX); hipFree(dmaxIterations);
    //
}
